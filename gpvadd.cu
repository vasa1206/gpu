#include "hip/hip_runtime.h"
#include "book.h"

#define N 5

__global__ void add (int *a, int *b, int *c) {
	int tid = blockIdx.x; // this is CPU zero, so we star at zero

	if (tid < N ) {
			c[tid] = a[tid] + b[tid];
	}
}

void addcpu( int *a, int *b, int *c ) {
	int tid = 0; // this is CPU zero, so we start at zero while (tid < N) {
	while(tid < N) {
        	c[tid] = a[tid] + b[tid];
		tid += 1; // we have one CPU, so we increment by one }
	}
}

int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// allocate GPU memory
	HANDLE_ERROR(hipMalloc((void **) &dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **) &dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **) &dev_c, N * sizeof(int)));

	//fill the arrays a & b on the CPU
	for(int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}

	// copy the arrays 'a' and 'b' to the GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

	add <<<N, 1>>> (dev_a, dev_b, dev_c);
	//addcpu(dev_a, dev_b, dev_c);

	// copy the array 'c' back from GPU to CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	for (int i=0; i < N; i++) {
		printf("%d + %d = c %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
