#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <book.h>

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

__global__ void kernel(void) {
	printf("in kernel\n");
}

int main(void) {
		int c;
		int *dev_c;
    		hipDeviceProp_t  prop;
    		int dev;

		HANDLE_ERROR( hipGetDeviceCount( &dev ) );

		printf(("ID of current CUDA device: %d\n", dev);

		memset(&prop, 0, sizeof(hipDeviceProp_t));

		prop.major = 1;
		prop.minor = 3;

		HANDLE_ERROR(hipChooseDevice( ((void **) &dev_c, sizeof(int)));

		add <<<1,1>>>(2, 7, dev_c);

		HANDLE_ERROR(hipMemcpy (&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
		printf(" 2+ 7 = %d\n", c);

		hipFree(dev_c);


	HANDLE_ERROR( hipGetDeviceCount( &count ) ); 
	for (int i=0; i< count; i++) {
        HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
        //Do something with our device's properties
	printf( " --- General Information for device %d ---\n", i ); printf( "Name: %s\n", prop.name );
	printf( "Compute capability: %d.%d\n", prop.major, prop.minor ); printf( "Clock rate: %d\n", prop.clockRate );
	printf( "Device copy overlap: " );
	if (prop.deviceOverlap)
		printf( "Enabled\n" ); 
		else
		printf( "Disabled\n" );
	printf( "Kernel execition timeout : " ); 
	if (prop.kernelExecTimeoutEnabled)
		printf( "Enabled\n" ); 
	else
            	printf( "Disabled\n" );
        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

             printf( "   --- MP Information for device %d ---\n", i );
           printf( "Multiprocessor count:  %d\n",
                       prop.multiProcessorCount );
           printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
           printf( "Registers per mp:  %d\n", prop.regsPerBlock );
           printf( "Threads in warp:  %d\n", prop.warpSize );
           printf( "Max threads per block:  %d\n",
                       prop.maxThreadsPerBlock );
           printf( "Max thread dimensions:  (%d, %d, %d)\n",
                       prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                       prop.maxThreadsDim[2] );
           printf( "Max grid dimensions:  (%d, %d, %d)\n",
                       prop.maxGridSize[0], prop.maxGridSize[1],
                       prop.maxGridSize[2] );
           printf( "\n" );
		} 
	}
