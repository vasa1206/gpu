#include "hip/hip_runtime.h"
#include "book.h"

#define N 10

__global__ void add( int *a, int *b, int *c ) {
	int tid = blockIdx.x; // handle the data at this index if (tid < N)
		if (tid < N) {
			printf("in kernal -- %d + %d = %d\n", a[tid], b[tid], c[tid]);
			c[tid] = a[tid] + b[tid];
		}
}

int main(void) {
	int a[N], b[N], c[N];

	int *dev_a, *dev_b, *dev_c;

	// allocate memory on GPU

	HANDLE_ERROR( hipMalloc((void**)&dev_a, N * sizeof(int)));

	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));

	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	// fill arrays 'a' and 'b' on CPU
	for (int i = 0; i<N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	
	// copy arrays 'a' and 'b' to GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));

	add<<<N, 1>>>(dev_a, dev_b, dev_c);

	// copy array 'c' from GPU to CPU
	//HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) );

	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	// display results
	for (int i=0; i<N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	//free the memory allocated on the GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
