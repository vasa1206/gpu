#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <book.h>

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

__global__ void kernel(void) {
	printf("in kernel\n");
}

int main(void) {
		int c;
		int *dev_c;
		
		HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

		add<1,1>>>(2, 7, dev_c);

		HANDLE_ERROE(cudaMencpy(&c, dev_c, sizeof(int), cudaMencpyDeviceToHost));

		printf("2 + 7 = %d\n", c);

		hipFree(dev_c);

		printf("Hello World!\n");

		return(0);
}
