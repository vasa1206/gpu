#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <book.h>

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

__global__ void kernel(void) {
	printf("in kernel\n");
}

int main(void) {
		int c;
		int *dev_c;
    		hipDeviceProp_t  prop;
    		int count;

		HANDLE_ERROR( hipGetDeviceCount( &count ) );

		HANDLE_ERROR(hipMalloc ((void **) &dev_c, sizeof(int)));

		add <<<1,1>>>(2, 7, dev_c);

		HANDLE_ERROR(hipMemcpy (&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
		printf(" 2+ 7 = %d\n", c);

		hipFree(dev_c);

		//kernel <<<1,1>>>();

		//printf("Hello World!\n");


	HANDLE_ERROR( hipGetDeviceCount( &count ) ); 
//	for (int i=0; i< count; i++) {
 //       HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
        //Do something with our device's properties
//		printf("%s\n", *prop.name);
//		} 
//	}
	return(0);
}
