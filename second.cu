
#include <hip/hip_runtime.h>
#include <stdio.h>

	__global__ void add(float *d_out, float *d_in) {
		int idx = threadIdx.x;
		float f = d_in[idx];
		d_out[idx] = f + f;
		printf("%i -> %f\n", idx, d_out[idx]);
	}

	__global__ void square(float *d_out, float *d_in) {
		int idx = threadIdx.x;
		float f = d_in[idx];
		d_out[idx] = f * f;
		printf("%i -> %f\n", idx, d_out[idx]);
	}

int main(int argc, char**argv) {
	printf("hello");

	const int ARRAY_SIZE= 1024;
	const int ARRAY_BYTES = ARRAY_SIZE *  sizeof(float);

	// declare and iniutialize CPU arrays
	float h_in[ARRAY_SIZE];
	float h_out[ARRAY_SIZE];

	for(int i=0   ; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}

	/// allocate GPU memory
	float *d_in;
	float  *d_out;
	hipMalloc((void **)&d_in, ARRAY_BYTES);
	hipMalloc((void **)&d_out, ARRAY_BYTES);

	// Copy array to GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// Launch the kernal
	square<<<1, ARRAY_SIZE>>> (d_out, d_in);
	//add<<<1, ARRAY_SIZE>>> (d_out, d_in);

	// Copy Results back to GPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// Cleanup
	hipFree(d_in);
	hipFree(d_out);
}
